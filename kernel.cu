﻿#include "hip/hip_runtime.h"


#include <algorithm>
#include <iostream>
#include <vector>
#include <string>
#include <time.h>
#include <chrono>
#include <thread>

using namespace std;
using namespace std::this_thread;
using namespace std::chrono_literals;
using std::chrono::system_clock;

// Helper function to generate random double between set intervals.
__device__ __host__ double doubleRand( double min , double max )
{
	double random = ( double ) rand() / RAND_MAX;
	return min + random * ( max - min );
}

class Racer
{
private:
	double		speed;
	double		pos;
	string		name;

public:
	__device__ __host__ Racer() : speed ( 0.0 ) , pos ( 0.0 ) , name( "" )
	{
		speed = doubleRand( 1.0 , 5.0 );
	}
	__device__ __host__ Racer( string name ) : speed( 0.0 ) , pos( 0.0 ) , name( name )
	{
		speed = doubleRand( 1.0 , 5.0 );
	}
	__device__ __host__ Racer( const Racer* other )
	{
		this->speed = other-> speed;
		this->pos = other->pos;
		this->name = other->name;
	}
	__device__ __host__ ~Racer() {}
	__device__ __host__ double getSpeed() const { return speed; }
	__device__ __host__ double getPos() const { return pos; }
	__device__ __host__ string getName() const { return name; }
	__device__ __host__ bool getWinnerState()
	{
		if( pos > 100.0 )
		{
			return true;
		}
		return false;
	}
	__device__ __host__ void calcNewPos()
	{
		pos += getSpeed();
	}
	__host__ void print()
	{
		cout << getName() << ": " << getPos() << "m\t" << "Speed: " << getSpeed() << "m/s" << endl;
	}

	bool operator < ( const Racer& other ) const
	{
		return this->getPos() < other.getPos();
	}
};

// Momentary printing
__host__ void raceStatus( Racer* list , int size )
{
	system( "CLS" );

	cout << "--- Stage Info ---" << endl;

	for( int i = 0; i < size; i ++ )
	{
		list[ i ].print();
	}
}

// Print the final results
__host__ void finalResults( Racer* list , int size )
{
	cout << "\n--- Final Results ---" << endl;

	vector< Racer > results;

	for( int j = 0; j < size; j++ )
	{
		results.push_back( list[ j ] );
	}

	std::sort( results.begin() , results.end() );

	for( int j = 0; j < size; j++ )
	{
		cout << j + 1;

		if( j == 10 || j == 11 || j == 12 )
		{
			cout << "th";
		}
		else
		{
			if( j % 10 < 3 )
			{
				if( j % 10 == 0 )
				{
					cout << "st";
				}
				else if( j % 10 == 1 )
				{
					cout << "nd";
				}
				else if( j % 10 == 2 )
				{
					cout << "rd";
				}
			}
			else
			{
				cout << "th";
			}
		}
		cout << " => " << results[ 99 - j ].getName() << "\t";

		if( (j + 1) % 4 == 0 )
		{
			cout << endl;
		}
	}

}

// GPU calculating new position of racers
__global__ void calcPos( Racer* list , int n )
{
	int i = threadIdx.x;

	list[ i ].calcNewPos();

	return;
}

// Helper function for using CUDA to calculate Racer positions.
hipError_t calcWithCuda( Racer* list , int size )
{
	Racer *dev_list = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on
	cudaStatus = hipSetDevice( 0 );
	if( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n" );
		goto Error;
	}

	// Allocate GPU buffers for list of Racers
	cudaStatus = hipMalloc( ( void** ) &dev_list , size * sizeof( Racer ) );
	if( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMalloc failed!\n" );
		goto Error;
	}

	// Copy input list from host to device
	cudaStatus = hipMemcpy( dev_list , list , size * sizeof( Racer ) , hipMemcpyHostToDevice );
	if( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMemcpy failed!\n" );
		goto Error;
	}

	// Launch a kernel on the GPU with one thread per Racer until we have a winner.
	while( true )
	{

		calcPos <<<1 , size >>> ( dev_list , size );

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if( cudaStatus != hipSuccess )
		{
			fprintf( stderr , "calcPos launch failed: %s\n" , hipGetErrorString( cudaStatus ) );
			goto Error;
		}
	
		// cudaDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if( cudaStatus != hipSuccess )
		{
			fprintf( stderr , "hipDeviceSynchronize returned error code %d after launching addKernel!\n" , cudaStatus );
			goto Error;
		}
	
		cudaStatus = hipMemcpy ( list , dev_list , size * sizeof( Racer ) , hipMemcpyDeviceToHost );
		if( cudaStatus != hipSuccess )
		{
			fprintf( stderr , "hipMemcpy failed!\n" );
			goto Error;
		}

		// Every time we return the racer list to the host, we must check if we have a winner.
		// Printing the race status at every iteration is entirely optional.
		raceStatus( list , 100 );

		for( int i = 0; i < size; i++ )
		{
			if( list[ i ].getWinnerState() )
			{
				// Invoke a false error which will leave the while loop.
				goto Error;
			}
		}


		sleep_for( 1s );
	}


	Error:
		hipFree( dev_list );

	return cudaStatus;

}

int main()
{
	srand ( time( NULL ) );

	Racer host_list[ 100 ];
	for( int k = 0; k < 100; k++ )
	{
		// For simplicity, I am just giving numbers as their names.
		host_list[ k ] = new Racer( to_string( k ) );
	}

	hipError_t cudaStatus = calcWithCuda( host_list , 100 );
	if( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "calcWithCuda failed!\n" );
		return -1;
	}

	finalResults( host_list , 100 );

	cudaStatus = hipDeviceReset();
	if( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipDeviceReset failed!\n" );
		return -1;
	}
	
	return 0;
}
